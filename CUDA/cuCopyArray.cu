#include "hip/hip_runtime.h"
/*********************************************************************
 * Copyrights (c) Marwan Abdellah. All rights reserved.
 * This code is part of my Master's Thesis Project entitled "High
 * Performance Fourier Volume Rendering on Graphics Processing Units
 * (GPUs)" and submitted to the Systems & Biomedical Engineering
 * Department, Faculty of Engineering, Cairo University.
 * Please, don't use or distribute without authors' permission.

 * File         : Volume
 * Author(s)    : Marwan Abdellah <abdellah.marwan@gmail.com>
 * Created      : April 2011
 * Description  :
 * Note(s)      :
 *********************************************************************/



#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ 
void copyArray_2D_float_kernel(float* devArrayOutput, float* devArrayInput, int nX)
{
 
    // Thread
    int xThreadIdx = threadIdx.x;
    int yThreadIdx = threadIdx.y;

    // Block Width & Height
    int blockWidth = blockDim.x;
    int blockHeight = blockDim.y;

    // Thread Index 2D  
    int xIndex = blockIdx.x * blockWidth + xThreadIdx;
    int yIndex = blockIdx.y * blockHeight + yThreadIdx;
    
    int index = (yIndex * nX) + xIndex;
    devArrayOutput[index] = 1.02145; 
}
    

    